#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "alloc3d_gpu.h"
#include "func.h"
#include "print.h"

__global__ void jacobi_v3dv1(double *d_u, double *d_uOld, double *d_f, int N, int N2, int iter_max, double frac, double delta2){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i>0 && i<N/2-1 && j>0 && j<N/2-1 && k>0 && k<N-1){ 
       	d_u[i*N2+j*N+k]	= frac*(d_uOld[(i-1)*N2+j*N+k]+d_uOld[(i+1)*N2+j*N+k]+d_uOld[i*N2+(j-1)*N+k]+d_uOld[i*N2+(j+1)*N+k]+d_uOld[i*N2+j*N+k-1]+d_uOld[i*N2+j*N+k+1]+delta2*d_f[i*N2+j*N+k]);
    }
}
__global__ void jacobi_v3dv2(double *d_u, double *d_uOld, double *d_f, int N, int N2, int iter_max, double frac, double delta2){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i>0 && i<N/2-1 && j>0 && j<N/2-1 && k>0 && k<N-1){ 
       	d_u[i*N2+j*N+k]	= frac*(d_uOld[(i-1)*N2+j*N+k]+d_uOld[(i+1)*N2+j*N+k]+d_uOld[i*N2+(j-1)*N+k]+d_uOld[i*N2+(j+1)*N+k]+d_uOld[i*N2+j*N+k-1]+d_uOld[i*N2+j*N+k+1]+delta2*d_f[i*N2+j*N+k]);
    }
}




int main(int argc, char *argv[]){
    
    int N = atoi(argv[1]);
    int iter_max = atoi(argv[2]);
    double start_T = atof(argv[3]);
    int output_type = 4;
    char *output_prefix = "poisson_j_gpu1";
    char *output_ext = "";
    char output_filename[FILENAME_MAX];


    int N2 = N * N;
    // Wake up gpu
    hipSetDevice(0);
    double *d_dummy;
    hipMalloc((void**)&d_dummy,0);
    hipSetDevice(1);
    hipMalloc((void**)&d_dummy,0);

    double *d_u, *d_uOld, *d_uSwap, *d_f, *d1_u, *d1_uOld, *d1_uSwap, *d1_f;
    double *h_u, *h_uOld, *h_uSwap, *h_f;
    double size = N * N * N * sizeof(double);
    // Pinning memory in host
    hipHostMalloc((void**)&h_u, size);
    hipHostMalloc((void**)&h_uOld, size);
    hipHostMalloc((void**)&h_uSwap, size);
    hipHostMalloc((void**)&h_f, size);

    // Initialization of the arrays
    u_init(h_u, N, N2, start_T); 
    u_init(h_uOld, N, N2, start_T); 
    u_init(h_uSwap, N, N2, start_T); 
    f_init(h_f, N, N2);

    // Device 0
    hipSetDevice(0);
    hipDeviceEnablePeerAccess(1, 0);

    // Device memory allocation 
    hipMalloc((void**)&d_u, size/2);
    hipMalloc((void**)&d_uOld, size/2);
    hipMalloc((void**)&d_uSwap, size/2);
    hipMalloc((void**)&d_f, size/2);

    // Copy initializationf from host to device
    hipMemcpy(d_u, h_u, size/2, hipMemcpyHostToDevice);
    hipMemcpy(d_uOld, h_uOld, size/2, hipMemcpyHostToDevice);
    hipMemcpy(d_uSwap, h_uSwap, size/2, hipMemcpyHostToDevice);
    hipMemcpy(d_f, h_f, size/2, hipMemcpyHostToDevice);

    // Device 1
    hipSetDevice(1);
    hipDeviceEnablePeerAccess(0, 0);

    // Device memory allocation 
    hipMalloc((void**)&d1_u, size/2);
    hipMalloc((void**)&d1_uOld, size/2);
    hipMalloc((void**)&d1_uSwap, size/2);
    hipMalloc((void**)&d1_f, size/2);

    // Copy initializationf from host to device
    hipMemcpy(d1_u, h_u + N/2, size/2, hipMemcpyHostToDevice);
    hipMemcpy(d1_uOld, h_uOld + N/2, size/2, hipMemcpyHostToDevice);
    hipMemcpy(d1_uSwap, h_uSwap + N/2, size/2, hipMemcpyHostToDevice);
    hipMemcpy(d1_f, h_f + N/2, size/2, hipMemcpyHostToDevice);
   

    // kernel settings
    dim3 blocksize(10,10,10);
    dim3 gridsize( ceil((double) N/(2*blocksize.x)),ceil((double) N/(2*blocksize.y)),ceil((double) N/(2*blocksize.z)) );
    
    // Jacobi max iterations loop in host
    double frac = 1.0/6.0;
    double delta2 = (2.0*2.0)/N2;
        // timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
    
    int it = 0;
    float elapsed=0, cycle;
    while(it < iter_max){
    
        hipEventRecord(start,0);
       




        hipSetDevice(0);
        
        d_uSwap = d_uOld;
        d_u = d_uOld;
        d_uOld = d_uSwap;

        jacobi_v3dv1<<<gridsize,blocksize>>>(d_u, d_uOld, d_f, N, N2, iter_max, frac, delta2);

        hipSetDevice(1);

        d1_uSwap = d1_uOld;
        d1_u = d1_uOld;
        d1_uOld = d1_uSwap;
        jacobi_v3dv2<<<gridsize,blocksize>>>(d1_u, d1_uOld, d1_f, N, N2, iter_max, frac, delta2);

        hipDeviceSynchronize();
        it++;
       
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&cycle, start, stop);
        elapsed += cycle;
    }

    printf("Operation finished!  GPU runtime (ms): %3.6f\n\n", elapsed);

    // Copy back to host
    hipMemcpy(h_u, d_u, size/2, hipMemcpyDeviceToHost);
    hipMemcpy(h_u + N/2, d1_u, size/2, hipMemcpyDeviceToHost);

    // dump  results if wanted
    switch(output_type) {
        case 0:
            // no output at all
            break;
        case 4:
            output_ext = ".vtk";
            sprintf(output_filename, "%s_%d%s", output_prefix, N, output_ext);
            fprintf(stderr, "Write VTK file to %s: ", output_filename);
            print_vtk(output_filename, N, h_u);
            break;
        default:
            fprintf(stderr, "Non-supported output type!\n");
            break;
    }

    //Free host and device memory    
    hipHostFree(h_f);
    hipHostFree(h_u);
    hipHostFree(h_uOld);
    hipHostFree(h_uSwap);
    
    hipHostFree(d_f);
    hipHostFree(d_u);
    hipHostFree(d_uOld);
    hipHostFree(d_uSwap);
   
    return(0); 
}

#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "alloc3d_gpu.h"
#include "func.h"
#include "print.h"

__global__ void jacobi_v1(double *d_u, double *d_uOld, double *d_f, int N, int N2, int iter_max, double frac, double delta2){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i>0 && i<N-1 && j>0 && j<N-1 && k>0 && k<N-1){
       	d_u[i*N2+j*N+k]	= frac*(d_uOld[(i-1)*N2+j*N+k]+d_uOld[(i+1)*N2+j*N+k]+d_uOld[i*N2+(j-1)*N+k]+d_uOld[i*N2+(j+1)*N+k]+d_uOld[i*N2+j*N+k-1]+d_uOld[i*N2+j*N+k+1]+delta2*d_f[i*N2+j*N+k]);
    }
}

int main(int argc, char *argv[]){
    
    int N = atoi(argv[1]);
    int iter_max = atoi(argv[2]);
    double start_T = atof(argv[3]);
    int output_type = 4;
    char *output_prefix = "poisson_j_gpu1";
    char *output_ext = "";
    char output_filename[FILENAME_MAX];


    int N2 = N * N;
    // Wake up gpu
    hipSetDevice(0);
    double *d_dummy;
    hipMalloc((void**)&d_dummy,0);

    double *d_u, *d_uOld, *d_uSwap, *d_f;
    double *h_u, *h_uOld, *h_uSwap, *h_f;
    double size = N * N * N * sizeof(double);

    // Device memory allocation 
    hipMalloc((void**)&d_u, size);
    hipMalloc((void**)&d_uOld, size);
    hipMalloc((void**)&d_uSwap, size);
    hipMalloc((void**)&d_f, size);

    // Pinning memory in host
    hipHostMalloc((void**)&h_u, size);
    hipHostMalloc((void**)&h_uOld, size);
    hipHostMalloc((void**)&h_uSwap, size);
    hipHostMalloc((void**)&h_f, size);

    // Initialization of the arrays
    u_init(h_u, N, N2, start_T); 
    u_init(h_uOld, N, N2, start_T); 
    u_init(h_uSwap, N, N2, start_T); 
    f_init(h_f, N, N2);

    // Copy initializationf from host to device
    hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);
    hipMemcpy(d_uOld, h_uOld, size, hipMemcpyHostToDevice);
    hipMemcpy(d_uSwap, h_uSwap, size, hipMemcpyHostToDevice);
    hipMemcpy(d_f, h_f, size, hipMemcpyHostToDevice);

    // kernel settings
    dim3 blocksize(10,10,10);
    dim3 gridsize( ceil((double) N/blocksize.x),ceil((double) N/blocksize.y),ceil((double) N/blocksize.z) );
    // Jacobi max iterations loop in host
    double frac = 1.0/6.0;
    double delta2 = (2.0*2.0)/N2;
        // timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
    
    int it = 0;
    hipEventRecord(start);
    while(it < iter_max){
        d_uSwap = d_uOld;
        d_u = d_uOld;
        d_uOld = d_uSwap;   
        jacobi_v1<<<gridsize,blocksize>>>(d_u, d_uOld, d_f, N, N2, iter_max, frac, delta2);
        hipDeviceSynchronize();
        
        it++;
    }
    hipEventRecord(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Operation finished!  GPU runtime (ms): %3.6f\n\n", milliseconds);

    // Copy back to host
    hipMemcpy(h_u, d_u, size, hipMemcpyDeviceToHost);

    // dump  results if wanted
    switch(output_type) {
        case 0:
            // no output at all
            break;
        case 4:
            output_ext = ".vtk";
            sprintf(output_filename, "%s_%d%s", output_prefix, N, output_ext);
            fprintf(stderr, "Write VTK file to %s: ", output_filename);
            print_vtk(output_filename, N, h_u);
            break;
        default:
            fprintf(stderr, "Non-supported output type!\n");
            break;
    }

    //Free host and device memory    
    hipHostFree(h_f);
    hipHostFree(h_u);
    hipHostFree(h_uOld);
    hipHostFree(h_uSwap);
    
    hipHostFree(d_f);
    hipHostFree(d_u);
    hipHostFree(d_uOld);
    hipHostFree(d_uSwap);
   
    return(0); 
}

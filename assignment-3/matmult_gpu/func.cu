#include "hip/hip_runtime.h"
// Asignment 3 Part 1 Matrix Multiplication
#include <hipblas.h> // for matmult_gpulib()
extern "C" {

    #include <stdio.h>
    #include <stdlib.h>
    #include <math.h> // for pow()
    #include <omp.h>
    #include <cblas.h> // for matmult_lib()

    #define stride_col 2
    #define stride_row 2
    #define stride 2
    #define BLOCK_SIZE 16

    /* Native CBLAS CPU implementation of matrix multiplication */
    void matmult_lib(int M, int N, int K, double *A, double *B, double *C) {
        double alpha = 1.0, beta = 0.0;
        cblas_dgemm(CblasRowMajor,CblasNoTrans,CblasNoTrans,M,N,K,alpha,A,K,B,N,beta,C, N);
    }

    /* part 1: sequential implementation in GPU (single thread) */
    __global__ void matmult_gpu1_kernel(int M, int N, int K, double* A, double *B, double* C) {
        double temp = 0.0;

        for (int i = 0; i < M; i++) {
            for (int j = 0; j < N; j++) {
                temp = 0.0;
                for (int k = 0; k < K; k++) {
                    temp += A[i*K + k] * B[k*N + j];
                }
                C[i*N + j] = temp;
            }
        }
    }

    void matmult_gpu1(int M, int N, int K, double* h_A, double *h_B, double* h_C) {
        double *d_A, *d_B, *d_C; // Device variables
        int size_A = M*K*sizeof(double);
        int size_B = N*K*sizeof(double);
        int size_C = N*M*sizeof(double);

        /* GPU: Allocate memory on device */
        hipMalloc((void**)&d_A, size_A);
        hipMalloc((void**)&d_B, size_B);
        hipMalloc((void**)&d_C, size_C);

        /* Copying data to device */
        hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

        /* MATRIX MULTIPLICATION */
        // Define grid and threads per block
        matmult_gpu1_kernel<<<1,1>>>(M, N, K, d_A, d_B, d_C);
        hipDeviceSynchronize();

        hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

        /* Freeing memory */
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    /* part 2: naive implementation in GPU (one thread per element in C) */
    __global__ void matmult_gpu2_kernel(int M, int N, int K, double* A, double* B, double* C) {
        double temp = 0.0;
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;

        if (i < M && j < N){ // ensure that the extra threads do not do any work
            for (int step = 0; step < K; step++) {
                temp += A[j*K + step] * B[step*N + i];
            }
            C[j*N + i] = temp;
        }
    }

    void matmult_gpu2(int M, int N, int K, double* h_A, double *h_B, double* h_C) {
        double *d_A, *d_B, *d_C; // Device variables
        int size_A = M*K*sizeof(double);
        int size_B = N*K*sizeof(double);
        int size_C = N*M*sizeof(double);

        /* GPU: Allocate memory on device */
        hipMalloc((void**)&d_A, size_A);
        hipMalloc((void**)&d_B, size_B);
        hipMalloc((void**)&d_C, size_C);

        /* Copying data to device */
        hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

        /* MATRIX MULTIPLICATION */
        // Define grid and threads per block
        dim3 blocksPerGrid(((N-1) / BLOCK_SIZE+1), ((M-1) / BLOCK_SIZE+1));
        dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);

        matmult_gpu2_kernel<<<blocksPerGrid,threadsPerBlock>>>(M, N, K, d_A, d_B, d_C);
        hipDeviceSynchronize();

        hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

        /* Freeing memory */
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    /* part 3: GPU (thread computes 2 elements of C) */
    __global__ void matmult_gpu3_kernel(int M, int N, int K, double* d_A, double* d_B, double* d_C) {
        double temp1 = 0.0;
        double temp2 = 0.0;

        int i = (blockIdx.x * blockDim.x + threadIdx.x);
        int j = (blockIdx.y * blockDim.y + threadIdx.y)*stride;

        if (i < M && j < N) {
            for (int k = 0; k < K; k++) {
                temp1 += d_A[(i)*K + k] * d_B[k*N + j];
                /* Below neighbour */
                if (j+1 < N) { // only if not end
                    temp2 += d_A[(i)*K + k] * d_B[k*N + (j+1)];
                }
                /* Right neighbour
                if (i+1 < N) { // only if not end
                    temp2 += d_A[(i+1)*K + k] * d_B[k*N + (j)];
                }  */
            }
            d_C[i*N + j] = temp1;
            if (j+1 < N) { // only if not end
                d_C[(i)*N + (j+1)] = temp2;
            }
        }
    }

    void matmult_gpu3(int M, int N, int K, double* h_A, double *h_B, double* h_C) {
        double *d_A, *d_B, *d_C; // Device variables
        int size_A = M*K*sizeof(double);
        int size_B = N*K*sizeof(double);
        int size_C = N*M*sizeof(double);

        /* GPU: Allocate memory on device */
        hipMalloc((void**)&d_A, size_A);
        hipMalloc((void**)&d_B, size_B);
        hipMalloc((void**)&d_C, size_C);

        /* Copying data to device */
        hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

        /* MATRIX MULTIPLICATION */
        // Define grid and threads per block
        dim3 blocksPerGrid(ceil(N/BLOCK_SIZE)+1, ceil(M/BLOCK_SIZE*stride)+1);
        dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);

        matmult_gpu3_kernel<<<blocksPerGrid,threadsPerBlock>>>(M, N, K, d_A, d_B, d_C);
        hipDeviceSynchronize();

        hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

        /* Freeing memory */
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    /* part 4: GPU (thread computes >2 elements of C) */
    __global__ void matmult_gpu4_kernel(int M, int N, int K, double* d_A, double* d_B, double* d_C) {
        double temp[stride_col][stride_row];
        int sc, sr;
        for (sc = 0; sc < stride_col; sc++){
            for (sr = 0; sr < stride_row; sr++){
                temp[sc][sr] = 0.0;
            }
        }

        int j = (blockIdx.x * blockDim.x + threadIdx.x)*stride_col;
        int i = (blockIdx.y * blockDim.y + threadIdx.y)*stride_row;

        if (i < M && j < N) {
            for (int k = 0; k < K; k++) {
                for (sc = 0; sc < stride_col; sc++) {
                    if (sc + j < N) {
                        for (sr = 0; sr < stride_row; sr++) {
                            if (sr + i < M) {
                                temp[sc][sr] += d_A[(i+sr)*K + k] * d_B[k*N + (j+sc)];
                            }
                        }
                    }
                }
            }
            for (sc = 0; sc < stride_col; sc++) {
                if (sc + j < N) {
                    for (sr = 0; sr < stride_row; sr++) {
                        if (sr + i < M) {
                            d_C[(i+sr)*N + (j + sc)] = temp[sc][sr];
                        }
                    }
                }
            }
        }
    }

    void matmult_gpu4(int M, int N, int K, double* h_A, double *h_B, double* h_C) {
        double *d_A, *d_B, *d_C; // Device variables
        int size_A = M*K*sizeof(double);
        int size_B = N*K*sizeof(double);
        int size_C = N*M*sizeof(double);

        /* GPU: Allocate memory on device */
        hipMalloc((void**)&d_A, size_A);
        hipMalloc((void**)&d_B, size_B);
        hipMalloc((void**)&d_C, size_C);

        /* Copying data to device */
        hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

        /* MATRIX MULTIPLICATION */
        // Define grid and threads per block
        dim3 blocksPerGrid(ceil(N/BLOCK_SIZE*stride_col)+1, ceil(M/BLOCK_SIZE*stride_row)+1);
        dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);

        matmult_gpu4_kernel<<<blocksPerGrid,threadsPerBlock>>>(M, N, K, d_A, d_B, d_C);
        hipDeviceSynchronize();

        hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        /* Freeing memory */
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    /* part 5: GPU (shared memory version) */

    #define BLOCKDIM 32

__global__ void kernelFunc_gpu5(int m,int n,int k,double *d_A,double *d_B,double *d_C);
void matmult_gpu5(int m, int n, int k, double *h_a, double *h_b, double *h_c){
  double* d_a, *d_b, *d_c;

  //Allocate memory on device
  hipMalloc((void**)&d_a, m*k*sizeof(double));
  hipMalloc((void**)&d_b, k*n*sizeof(double));
  hipMalloc((void**)&d_c, m*n*sizeof(double));

  //Copy matrices to device memory
  hipMemcpy(d_a, h_a, m*k*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, k*n*sizeof(double), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(BLOCKDIM,BLOCKDIM);
  dim3 blocks((n-1)/BLOCKDIM+1, (m-1)/BLOCKDIM+1);

  //Call kernel
  kernelFunc_gpu5<<<blocks,threadsPerBlock>>>(m, n, k, d_a, d_b, d_c);
  hipDeviceSynchronize();

  //Copy result to host memory
  hipMemcpy(h_c, d_c, m*n*sizeof(double), hipMemcpyDeviceToHost);

  //Free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

    /* part 6: DGEMM function for GPUs, NVIDIA */
    void matmult_gpulib(int M, int N, int K, double* A, double *B, double* C) {

        /* Declare handle and initialize cublas */
        hipblasHandle_t handle;
        hipblasStatus_t status = hipblasCreate(&handle);
        if (status != HIPBLAS_STATUS_SUCCESS) { // check if init successful
            printf("Error: Initialization error CUBLAS. \n");
            exit(1);
        }

        double alpha = 1.0; // no prefactor
        double beta = 0.0; // C matrix not involved

    
        status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, A, M, B, N, &beta, C, K);
        if (status != HIPBLAS_STATUS_SUCCESS) { // check no errors are outputed in the execution
            printf("Error: Execution error CUBLAS. \n");
            exit(1);
        }

        /* Destroy handle and free memory */
        status = hipblasDestroy(handle);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("Error: Error destroying CUBLAS handle. \n");
            exit(1);
        }
    }


__global__ void kernelFunc_gpu5(int m,int n,int k,double *d_A,double *d_B,double *d_C);
void matmult_gpu5(int m, int n, int k, double *h_a, double *h_b, double *h_c){
  double* d_a, *d_b, *d_c;

  //Allocate memory on device
  hipMalloc((void**)&d_a, m*k*sizeof(double));
  hipMalloc((void**)&d_b, k*n*sizeof(double));
  hipMalloc((void**)&d_c, m*n*sizeof(double));

  //Copy matrices to device memory
  hipMemcpy(d_a, h_a, m*k*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, k*n*sizeof(double), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(BLOCKDIM,BLOCKDIM);
  dim3 blocks((n-1)/BLOCKDIM+1, (m-1)/BLOCKDIM+1);

  //Call kernel
  kernelFunc_gpu5<<<blocks,threadsPerBlock>>>(m, n, k, d_a, d_b, d_c);
  hipDeviceSynchronize();

  //Copy result to host memory
  hipMemcpy(h_c, d_c, m*n*sizeof(double), hipMemcpyDeviceToHost);

  //Free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

__global__ void kernelFunc_gpu5(int m, int n, int k, double *d_a, double* d_b, double* d_c){

    //Block indecies
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    //Row, Col inside the Block
    int row = threadIdx.y;
    int col = threadIdx.x;

    //Sub matrix C
    double *Csub = &d_c[blockRow*BLOCKDIM*n+blockCol*BLOCKDIM];

    //Inter mediate sum for current thread
    double intSum = 0.0;

    for(int i=0; i< (k/BLOCKDIM); i++){
      //Get Submatrix of A
      double *Asub = &d_a[blockRow*BLOCKDIM*k+BLOCKDIM*i];
      double *Bsub = &d_b[n*BLOCKDIM*i+blockCol*BLOCKDIM];

      __shared__ double As[BLOCKDIM][BLOCKDIM];
      __shared__ double Bs[BLOCKDIM][BLOCKDIM];

      As[row][col] = Asub[row*k+col];
      Bs[row][col] = Bsub[row*n+col];

      __syncthreads();

      for(int j=0; j<BLOCKDIM;j++){
        intSum += As[row][j]*Bs[j][col];
      }

      __syncthreads();
    }
    Csub[row*n+col] = intSum;
}
